#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>

#include "error.cuh"

const double EPSILON = 1.0e-15;
const double a = 1.23;
const double b = 2.34;
const double c = 3.57;

// 核函数的返回值必须是 void
// 必须要__global__， 可以使用static
// 不可以使用可变参数
// 非指针变量对每一个线程可见
// 除非使用统一内存，否则传入的指针必须是设备内存指针
__global__ void add(const double *x, const double *y, double *z, int N);
__device__ void add_device(double x, double y, double *z);
void check(const double *z, int N);

int main()
{
    const int N = 100000000;
    const int M = sizeof(double) * N;
    double *h_x = (double*) malloc(M);
    double *h_y = (double*) malloc(M);
    double *h_z = (double*) malloc(M);

    for (int n = 0; n < N; ++n) {
        h_x[n] = a;
        h_y[n] = b;
    }

    double *d_x, *d_y, *d_z;
    CHECK(hipMalloc(&d_x, M));
    hipMalloc(&d_y, M);
    hipMalloc(&d_z, M);
    hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice);

    const int block_size = 128;

    // 即使不能够整除也能够执行
    const int grid_size = (N - 1) / block_size + 1;
    add<<<grid_size, block_size>>>(d_x, d_y, d_z, N);

    // 检查核函数是否出错
    // 例如将线程数量设的足够大就会报错
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());

    hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost);
    check(h_z, N);

    free(h_x);
    free(h_y);
    free(h_z);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    return 0;
}


__global__ void add(const double *x, const double *y, double *z, int N) {
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n < N)
        add_device(x[n], y[n], &z[n]);
}

__device__ void add_device(const double x, const double y, double *z) {
    *z = x + y;
}

void check(const double *z, const int N) {
    bool has_error = false;
    for (int n = 0; n < N; ++n) {
        if (fabs(z[n] - c) > EPSILON) {
            has_error = true;
        }
    }
    printf("%s\n", has_error ? "Has errors" : "No errors");
}