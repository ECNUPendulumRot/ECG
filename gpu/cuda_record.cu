#include <cmath>
#include <cstdio>

#include "common.cuh"

int main()
{
    // A common method to record program execution time
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    CHECK(hipEventRecord(start));
    hipEventQuery(start);

    // Some Code blocks

    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));
    float elapsed_time;
    CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("Elapsed Time: %f ms\n", elapsed_time);
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));

    return 0;
}