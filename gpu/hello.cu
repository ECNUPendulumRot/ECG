
#include <hip/hip_runtime.h>
#include <cstdio>


__global__ void hello_from_gpu()
{
    const int bid = blockIdx.x;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    printf("Hello World from block %d and thread (%d, %d)!\n", bid, tx, ty);
}


int main()
{
    const dim3 block_size(2, 4);
    hello_from_gpu<<<1, block_size>>>();
    hipDeviceSynchronize();
    return 0;
}